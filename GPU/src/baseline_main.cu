#include <iostream>
#include "ctime"
#include "fstream"
#include "vector"
#include "host_common.h"
#include "device_common.cuh"
#include "baseline_sampling.cuh"
#include <hip/hip_runtime.h>
int main(int argc, char **argv) {
    if (argc != 3) {
        printf("please run this program by the following parameter: sample_number filePath\n");
        return 0;
    }
    //check gpu

    check_GPU();

    int sample_number = atoi(argv[1]);
    std::string filename =  argv[2];

    //read point
    std::ifstream fin(filename);
    if (!fin.is_open()) {
        std::cout << "file not exist" << std::endl;
        return 0;
    }
    std::vector <Point> point_data;
    int count = 0;
    if (fin.is_open()) {
        float xx, yy, zz;
        while (fin >> xx >> yy >> zz) {
            point_data.push_back({xx, yy, zz});
            count++;
        }
    }
    fin.close();
    const int point_data_size = point_data.size();


    float (*coordinates) = new float[point_data_size * 3];
    float (*result_cpu) = new float[sample_number * 3];
    float * d_coord;
    float * result;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for(int i = 0 ;i < point_data_size ; i++){
        coordinates[i*3] = point_data[i].pos[0];
        coordinates[i*3+1] = point_data[i].pos[1];
        coordinates[i*3+2] = point_data[i].pos[2];
    }
    //warmup
    warmup<<<1, 1>>>();
    hipDeviceSynchronize();


    hipEventRecord(start);

    hipMalloc((void **) &d_coord, (point_data_size)*sizeof(float)*3);
    hipMalloc((void **) &result, (sample_number)*sizeof(float)*3);
    hipMemcpy(d_coord,coordinates,point_data_size *sizeof(float )*3 ,hipMemcpyHostToDevice);
    farthest_point_sampling(point_data_size,sample_number,d_coord,result);

    hipEventRecord(stop);
    hipMemcpy(result_cpu,result, sample_number * 3 * sizeof(float), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    hipError_t err;
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    //read point
    std::ofstream fout("baseline.txt");
    if (!fout.is_open()) {
        std::cout << "file failed to open" << std::endl;
        return 0;
    }
    for(int i = 0 ;i < sample_number ;i ++){
        fout << result_cpu[i*3] << " " << result_cpu[i*3+1] << " " << result_cpu[i*3+2] << std::endl;
    }

    fout.close();

    std::cout << "Report:" << std::endl;
    std::cout << "    Type   :baseline(GPU)" << std::endl;
    std::cout << "    Points :" << point_data_size<< std::endl;
    std::cout << "    NPoint :" << sample_number << std::endl;
    std::cout << "    RunTime:" << milliseconds<< "ms" << std::endl;
    std::cout << "    Param  :" << filename << std::endl;
    std::time_t time_result = std::time(NULL);
    std::cout << "  Timestamp:" << std::asctime(std::localtime(&time_result)) << std::endl;

    hipFree(d_coord);
    hipFree(result);
    free(coordinates);
    free(result_cpu);
    return 0;

}



#include "hip/hip_runtime.h"
#include "device_common.cuh"

__global__ void warmup(){
    return;
}


void check_GPU(){
    hipError_t cudaStatus;
    int num = 0;
    cudaStatus = hipGetDeviceCount(&num);
    std::cout << "Number of GPU: " << num << std::endl;
    hipDeviceProp_t prop;
    if (num > 0) {
        hipGetDeviceProperties(&prop, 0);
        std::cout << "Device: " << prop.name << std::endl;
    }
}

__device__ void __update(float *__restrict__ dists, int *__restrict__ dists_i, int idx1, int idx2){
    const float v1 = dists[idx1], v2 = dists[idx2];
    const int i1 = dists_i[idx1], i2 = dists_i[idx2];
    dists[idx1] = max(v1, v2);
    dists_i[idx1] = v2 > v1 ? i2 : i1;
}

__device__ void merge(float *__restrict__ dists, int *__restrict__ dists_i,int tid, int block_size){
    if (block_size >= 4096) {
        if (tid < 2048) {
            __update(dists, dists_i, tid, tid + 2048);
        }
        __syncthreads();
    }
    if (block_size >= 2048) {
        if (tid < 1024) {
            __update(dists, dists_i, tid, tid + 1024);
        }
        __syncthreads();
    }

    if (block_size >= 1024) {
        if (tid < 512) {
            __update(dists, dists_i, tid, tid + 512);
        }
        __syncthreads();
    }

    if (block_size >= 512) {
        if (tid < 256) {
            __update(dists, dists_i, tid, tid + 256);
        }
        __syncthreads();
    }
    if (block_size >= 256) {
        if (tid < 128) {
            __update(dists, dists_i, tid, tid + 128);
        }
        __syncthreads();
    }
    if (block_size >= 128) {
        if (tid < 64) {
            __update(dists, dists_i, tid, tid + 64);
        }
        __syncthreads();
    }
    if (block_size >= 64) {
        if (tid < 32) {
            __update(dists, dists_i, tid, tid + 32);
        }
        __syncthreads();
    }
    if (block_size >= 32) {
        if (tid < 16) {
            __update(dists, dists_i, tid, tid + 16);
        }
        __syncthreads();
    }

    if (block_size >= 16) {
        if (tid < 8) {
            __update(dists, dists_i, tid, tid + 8);
        }
        __syncthreads();
    }

    if (block_size >= 8) {
        if (tid < 4) {
            __update(dists, dists_i, tid, tid + 4);
        }
        __syncthreads();
    }
    if (block_size >= 4) {
        if (tid < 2) {
            __update(dists, dists_i, tid, tid + 2);
        }
        __syncthreads();
    }
    if (block_size >= 2) {
        if (tid < 1) {
            __update(dists, dists_i, tid, tid + 1);
        }
        __syncthreads();
    }
}

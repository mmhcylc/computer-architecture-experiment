#include "hip/hip_runtime.h"
//
// Created by hanm on 22-7-5.
//
#include <iostream>
#include "ctime"
#include "fstream"
#include "device_common.cuh"
#include "algorithm"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include "kdtree.cuh"

int main(int argc, char **argv) {
    if (argc != 4) {
        printf("please run this program by the following parameter: kdtree_high sample_number filePath\n");
        return 0;
    }

    //check gpu

    check_GPU();

    int kd_high = atoi(argv[1]);
    int sample_number = atoi(argv[2]);
    std::string filename =  argv[3];

    //read point
    std::ifstream fin(filename);
    if (!fin.is_open()) {
        std::cout << "file not exist" << std::endl;
        return 0;
    }
    thrust::host_vector<float3> point_data;
    int count = 0;
    if (fin.is_open()) {
        float xx, yy, zz;
        while (fin >> xx >> yy >> zz) {
            point_data.push_back({xx, yy, zz});
            count++;
        }
    }
    fin.close();
    const int point_data_size = point_data.size();

    float3 result_cpu[sample_number];
    int bucketSize = 1 << kd_high;

    hipEvent_t start, build_end, stop;
    hipEventCreate(&start);
    hipEventCreate(&build_end);
    hipEventCreate(&stop);
    //warmup
    warmup<<<1, 1>>>();
    hipDeviceSynchronize();

    //build

    hipEventRecord(start);

    thrust::device_vector<float3> dPoints=point_data;
    float3 * ptr = thrust::raw_pointer_cast(&dPoints[0]);

    float3 * up;
    float3 * down;
    float3 * result;

    thrust::device_vector<int> bucketIndexVector(bucketSize);
    thrust::device_vector<int> bucketLengthVector(bucketSize);

    thrust::fill(bucketIndexVector.begin(), bucketIndexVector.end(), 0);
    thrust::fill(bucketLengthVector.begin(), bucketLengthVector.end(), point_data_size);

    int * bucketIndex = thrust::raw_pointer_cast(&bucketIndexVector[0]);
    int * bucketLength = thrust::raw_pointer_cast(&bucketLengthVector[0]);

    hipMalloc((void **)&up, bucketSize*sizeof(float3));
    hipMalloc((void **)&down, bucketSize*sizeof(float3));
    hipMalloc((void **)&result, sample_number*sizeof(float3));

    buildKDTree(bucketIndex, bucketLength, ptr, kd_high, up, down, point_data_size);

#ifdef  DEBUG_GG
    thrust::host_vector<int>cpu_bucketLength(bucketSize);
    thrust::copy(bucketLengthVector.begin(), bucketLengthVector.end(), cpu_bucketLength.begin());
    for(const auto & leng: cpu_bucketLength){
        printf("len: %d\n", leng);
    }
#endif
    hipEventRecord(build_end);
    //fps
    sample(bucketIndex, bucketLength, ptr, point_data_size, bucketSize, up, down, sample_number, result);

    hipEventRecord(stop);

    hipMemcpy((void *)result_cpu,(void *)result, sample_number*sizeof(float3), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);

    //read point
    std::ofstream fout("kdtree.txt");
    if (!fout.is_open()) {
        std::cout << "file failed to open" << std::endl;
        return 0;
    }
    for(const auto& point : result_cpu){
        fout << point.x << " " << point.y << " " << point.z << std::endl;
    }

    fout.close();

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float milliseconds_build = 0;
    hipEventElapsedTime(&milliseconds_build, start, build_end);

    float milliseconds_sample = 0;
    hipEventElapsedTime(&milliseconds_sample, build_end, stop);

    std::cout << "Report:" << std::endl;
    std::cout << "    Type   :kdline(GPU) high:" << kd_high << std::endl;
    std::cout << "    Points :" << point_data_size<< std::endl;
    std::cout << "    NPoint :" << sample_number << std::endl;
    std::cout << "    RunTime:" << milliseconds << "ms" << std::endl;
    std::cout << "    BuildTime:" << milliseconds_build << "ms(" << milliseconds_build*100/milliseconds << "%)" << std::endl;
    std::cout << "    SampleTime:" << milliseconds_sample << "ms(" << milliseconds_sample*100/milliseconds << "%)" << std::endl;
    std::cout << "    Param  :" << filename << std::endl;
    std::time_t time_result = std::time(NULL);
    std::cout << "  Timestamp:" << std::asctime(std::localtime(&time_result)) << std::endl;

    hipFree(up);
    hipFree(down);
    hipFree(result);

    return 0;
}